#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)
    
__global__ void scan(float * input, float * output, float* blocksum, int len) 
{
    //@@ Functionality of the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
	__shared__ float XY[2*BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    unsigned int start = blockIdx.x*blockDim.x;
    XY[t] = start + t > len ? 0.0 : input[start + t];
    XY[blockDim.x+t] = start + blockDim.x+t > len ? 0.0 : input[start + blockDim.x+t];
	for (int stride = 1;stride <= BLOCK_SIZE; stride *= 2) 
	{
        int index = (threadIdx.x+1)*stride*2 - 1;
        if(index < 2*BLOCK_SIZE)
                         XY[index] += XY[index-stride];
        __syncthreads();

    }
	
	for (int stride = BLOCK_SIZE/2; stride > 0; stride /= 2) 
	{
        __syncthreads();
        int index = (threadIdx.x+1)*stride*2 - 1;
        if(index+stride < 2*BLOCK_SIZE) 
		 {
            XY[index + stride] += XY[index];
		 }
	}
	
	__syncthreads();
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) 
	{
        output[i] = XY[threadIdx.x];
        if ((i+1)%blockDim.x == 0) blocksum[i/blockDim.x]=output[i];
	}	
	
}

__global__ void helper(float * output, float * blocksum, int len) 
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len)
	{
	        for (int j=0; j < i/blockDim.x; j++)
            output[i] += blocksum[j];
    }
}
    
int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
	float * deviceTemp;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*) malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
    //@@ deviceTemp
	wbCheck(hipMalloc((void**)&deviceTemp,  numElements*sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements*sizeof(float)));
	//@@ deviceTemp
	wbCheck(hipMemset(deviceTemp, 0, numElements/BLOCK_SIZE * sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid((numElements + BLOCK_SIZE - 1)/BLOCK_SIZE, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    wbLog(TRACE, "DimGrid ", (numElements + BLOCK_SIZE - 1)/BLOCK_SIZE);
    wbLog(TRACE, "DimBlock ", BLOCK_SIZE);
	//@@ Initialized grid and block dimensions here
	
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
	scan<<<DimGrid,DimBlock>>>(deviceInput, deviceOutput, deviceTemp, numElements);
    helper<<<DimGrid,DimBlock>>>(deviceOutput, deviceTemp, numElements);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
	hipFree(deviceTemp);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

